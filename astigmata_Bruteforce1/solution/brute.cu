#include "hip/hip_runtime.h"
// 2008 andrewl

#undef TIMESHIT
//#define TIMESHIT 1

#include <windows.h>

#include <stdio.h>

#include <hip/hip_runtime.h>

typedef struct _STATE
{
    DWORD eax;
    DWORD ebx;
    CHAR CF;
} STATE, *PSTATE;

__global__ void bruter(STATE *a, int N)
{
    int idx;
    ULONG i;
    ULONG shamt;
    ULONG temp;

    idx = blockIdx.x*blockDim.x + threadIdx.x;

//    if(idx < N)
//    {
    ULONG eax = a[idx].eax;
    ULONG ebx = a[idx].ebx;
    UCHAR CF = a[idx].CF;

    for(i=0; i<4000000; ++i)
    {
        // emulate RCR EAX, AL

        shamt = eax & 0x1F;

        if(shamt)
        {
            ULONG left = 0, right, ceffect;

            right = eax >> shamt;

            shamt = 32 - shamt;
            ceffect = CF << shamt;

            if(++shamt < 32)
                left = eax << shamt;

            eax = left | ceffect | right;          
        }

        eax += ebx;
        eax += 0x11223344;

        CF = 0;
        if(eax < 0x11223344)
            CF = 1;

        temp = eax;
        eax = ebx;
        ebx = temp;
    }

    a[idx].eax = eax;
    a[idx].ebx = ebx;
    a[idx].CF = CF;
//    }
}

UCHAR g_trial_digits[8]={0};

__forceinline DWORD next_eax()
{
    // ugly, but for speed

    DWORD ret =     g_trial_digits[0] |
                    (g_trial_digits[1] << 4) |
                    (g_trial_digits[2] << 8) |
                    (g_trial_digits[3] << 12) |
                    (g_trial_digits[4] << 16) |
                    (g_trial_digits[5] << 20) |
                    (g_trial_digits[6] << 24) |
                    (g_trial_digits[7] << 28);

    g_trial_digits[0]++;
    if(g_trial_digits[0] <= 9)
        return ret;
    g_trial_digits[0] = 0;
    g_trial_digits[1]++;
    if(g_trial_digits[1] <= 9)
        return ret;
    g_trial_digits[1] = 0;
    g_trial_digits[2]++;
    if(g_trial_digits[2] <= 9)
        return ret;
    g_trial_digits[2] = 0;
    g_trial_digits[3]++;
    if(g_trial_digits[3] <= 9)
        return ret;
    g_trial_digits[3] = 0;
    g_trial_digits[4]++;
    if(g_trial_digits[4] <= 9)
        return ret;
    g_trial_digits[4] = 0;
    g_trial_digits[5]++;
     if(g_trial_digits[5] <= 9)
        return ret;
    g_trial_digits[5] = 0;
    g_trial_digits[6]++;
    if(g_trial_digits[6] <= 9)
        return ret;
    g_trial_digits[6] = 0;
    g_trial_digits[7]++;
    if(g_trial_digits[7] <= 9)
        return ret;
    g_trial_digits[7] = 0; // around the corner!
    return ret;
}

unsigned __int64 hexStrToUint64(const char * hex_string)
{
    unsigned __int64 result = 0;
    int len=0, i=0;
    unsigned char value;

    if(hex_string[0]=='0' && hex_string[1]=='x')
        hex_string += 2;

    // find strlen
    len=-1;
    while(hex_string[++len]!='\0');

    if(len > 8)
        return 0xFFFFFFFF;

    for(i=0; i<len; i++) {
        char temp = hex_string[i];

        if((temp>=0x30) && (temp <=0x39)) // [0-9]
            value = temp - 0x30;
        else if(temp>=0x41 && temp<=0x46) // [A-F]
            value = temp - 0x37;
        else if(temp>=0x61 && temp<=0x66) // [A-Fa-f]
            value = temp - 0x57;
        else
            return 0xFFFFFFFF;

        result = result | (ULONG)(value << ((len-i-1) * 4));
    }

    return result;
}

void print_state(STATE s)
{
    printf("{0x%08X,0x%08X,%d}", s.eax, s.ebx, s.CF);
}

HANDLE g_hEvent0 = 0; // thread signals that he is alive
HANDLE g_hEvent1 = 0; // main signals that status message is awaiting
HANDLE g_hThread = 0;
CHAR g_LastStatusMessage[256];

DWORD WINAPI StatusReporter(LPVOID param)
{
    CHAR urlbuff[512];

    SetEvent(g_hEvent0);

    while(1)
    {
        printf("THREAD: waiting for work to report...\n");

        WaitForSingleObject(g_hEvent1, INFINITE);

        printf("THREAD: uploading message: %s\n", g_LastStatusMessage);

        sprintf(urlbuff, "http://andrewl.us/astigmata/?liwfEIJFkefji3=%s", g_LastStatusMessage);

        if(URLDownloadToFileA(0, urlbuff, "c:\\temp.txt", 0, 0) != S_OK)
            printf("THREAD: fuck man, upload failed!\n");
    }
}


int main(int argc, char * argv[])
{
    #define BLOCKSIZE 512
    #define NBLOCKS 24
    #define BATCHSIZE (BLOCKSIZE*NBLOCKS)

    // time measurement stuff
    #ifdef TIMESHIT
    FILETIME t0, t1;
    ULONGLONG s1;
    ULONGLONG s0;
    ULONGLONG diff;
    float difff;
    #endif

    // other vars
    hipError_t err;

    // memories
    PSTATE trials_host = 0; 
    PSTATE trials_device = 0;
    PSTATE results = 0;

    // parse args
    if(argc > 1)
    {
        ULONG init = hexStrToUint64(argv[1]);
    
        for(int i=0; i<8; ++i)
            g_trial_digits[i] = ((0x0F << (4*i)) & init) >> (4*i);
    }

    size_t size = BATCHSIZE * sizeof(STATE);

    // create event to signal thread
    g_hEvent0 = CreateEvent(0, 0, 0, 0); // no security, auto-reset, initially 0, no name
    g_hEvent1 = CreateEvent(0, 0, 0, 0); // no security, auto-reset, initially 0, no name
    HANDLE g_hThread = CreateThread(0, 0, StatusReporter, 0, 0, 0);
    WaitForSingleObject(g_hEvent0, INFINITE);

    // allocate arrays on host
    trials_host = (PSTATE)malloc(size);
    memset(trials_host, 0, size);
    results = (PSTATE)malloc(size);

    // allocate array on device 
    err = hipMalloc((PVOID *)&trials_device, size);
    if(err != hipSuccess)
        goto cleanup;

    while(1)
    {
        for(int i=0; i<BATCHSIZE; i++) 
            trials_host[i].eax = next_eax();

        //
        // signal thread to upload results
        //
        sprintf(    g_LastStatusMessage,
                    "searching [%08X, %08X]",
                    trials_host[0].eax, 
                    trials_host[BATCHSIZE-1].eax
                );

        SetEvent(g_hEvent1);

        // copy data from host to device
        err = hipMemcpy(trials_device, trials_host, size, hipMemcpyHostToDevice);
        if(err != hipSuccess)
            goto cleanup;

        #ifdef TIMESHIT
        GetSystemTimeAsFileTime(&t0);
        #endif

        for(INT i=0; i<25; ++i)
        {
            bruter <<< NBLOCKS, BLOCKSIZE >>> ((PSTATE)trials_device, BATCHSIZE);
   
            err = hipDeviceSynchronize();
            if(err != hipSuccess)
                goto cleanup;
            
            printf("called kernel %d\r", i);
        }

        err = hipMemcpy(results, trials_device, size, hipMemcpyDeviceToHost);
        if(err != hipSuccess)
            goto cleanup;

        #ifdef TIMESHIT
        GetSystemTimeAsFileTime(&t1);
        s1 = ((ULONGLONG)t1.dwHighDateTime << 32) | t1.dwLowDateTime;
        s0 = ((ULONGLONG)t0.dwHighDateTime << 32) | t0.dwLowDateTime;
        diff = (s1 - s0);
        difff = diff / (float)10000000;
        #endif

        // check results
        for (INT i=0; i<BATCHSIZE; i++)
        {
            /*
            printf("result: ");
            print_state(trials_host[i]);
            printf("->");
            print_state(results[i]);
            printf("\n");
            */

            if((results[i].eax ^ results[i].ebx) == 0xD5446474 )
            //if((results[i].eax ^ results[i].ebx) == 0xd3c7896e )
            {
                sprintf(    g_LastStatusMessage,
                            "!!! answer: 0x%08X",
                            trials_host[i].eax
                       );

                SetEvent(g_hEvent1);

                printf(g_LastStatusMessage);

                Sleep(0xFFFFFFFF);

                goto cleanup;
            }
        }

        #ifdef TIMESHIT
        printf("batch took %f seconds, meaning %f seconds per trial\n", difff, difff/(float)BATCHSIZE);
        #endif
    }

    cleanup:

    if(err != hipSuccess)
        printf("err: %s\n", hipGetErrorString(err));

    if(g_hThread)
        CloseHandle(g_hThread);

    if(g_hEvent0)
        CloseHandle(g_hEvent0);

    if(g_hEvent1)
        CloseHandle(g_hEvent1);

    if(trials_host)
        free(trials_host);

    if(trials_device)
        hipFree(trials_device); 
}

